
#include <hip/hip_runtime.h>
#define W 500
#define H 500
#define D 500
#define TX 32
#define TY 32
#define TZ 32

int divUp(int a, int b){return (a+b-1)/b;}
__device__ float distance(int c,int r, int s ,float3 pos)
{
	return sqrtf((c-pos.x)*(c-pos.x)+(r-pos.y)*(r-pos.y)+(s-pos.z)*(s-pos.z));

}

__global__
void distanceKernel(float *d_out, int w, int h, int d,float3 pos)
{
	const int c = blockIdx.x*blockDim.x+threadIdx.x;
	const int r = blockIdx.y*blockDim.y+threadIdx.y;
	const int s = blockIdx.z*blockDim.z+threadIdx.z;
	const int i = c+r*w+s*w*h;
	if((c>=w) || (r>=h) || (s>=d)) return;

	d_out[i]=distance(c,r,s,pos);
}

int main()
{
	float *out=(float *)calloc(W*H*D, sizeof(float));
	float *d_out;
	hipMalloc(&d_out, W*H*D*sizeof(float));
	const float3 pos={0.0f,0.0f,0.0f};
	const dim3 blockSize(TX,TY,TZ);
	const int bx = divUp(W, TX);
	const int by = divUp(H, TY);
	const int bz = divUp(D, TZ);
	const dim3 gridSize(bx,by,bz);
	distanceKernel<<<gridSize,blockSize>>>(d_out, W,H,D,pos);
	hipMemcpy(out, d_out, W*H*D*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_out);
	free(out);
	return 0;
}
