
#include <hip/hip_runtime.h>
#define W 500
#define H 500
#define D 500
#define TX 32
#define TY 32
#define TZ 32

__global__
void distanceKernel(float *d_out, int w, int h, int d,float3 pos)
{
	const int c = blockIdx.x*blockDim.x+threadIdx.x;
	const int r = blockIdx.y*blockDim.y+threadIdx.y;
	const int s = blockIdx.z*blockDim.z+threadIdx.z;
	const int i = c+r*w+s*w*h;
	if((c>=w) || (r>=h) || (s>=d)) return;

	d_out[i]=sqrtf((c-pos.x)*(c-pos.x)+(r-pos.y)*(r-pos.y)+(s-pos.z)*(s-pos.z));
}

int main()
{
	float *out=(float *)calloc(W*H*D, sizeof(float));
	float *d_out;
	hipMalloc(&d_out, W*H*D*sizeof(float));
	const float3 pos={0.0f,0.0f,0.0f};
	const dim3 blockSize(TX,TY,TZ);
	const int bx = (W+TX-1)/TX;
	const int by = (H+TY-1)/TY;
	const int bz = (D+TZ-1)/TZ;
	const dim3 gridSize(bx,by,bz);
	distanceKernel<<<gridSize,blockSize>>>(d_out, W,H,D,pos);
	hipMemcpy(out, d_out, W*H*D*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_out);
	free(out);
	return 0;
}
