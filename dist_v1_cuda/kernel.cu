
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 64 // Specify a constant value for array length.
#define TPB 32 // Threads per block

// A scaling function to convert integers 0,1,...,N-1
// to evenly spaced floats ranging from 0 to 1.
__device__ float scale(int i, int n)
{
  return ((float)i) / (n - 1);
}

// Compute the distance between 2 points on a line.
__device__ float distance(float x1, float x2)
{
  return sqrt((x2 - x1)*(x2 - x1));
}

__global__ void distanceKernel(float *d_out, float ref, int len)
{
	const int i = blockIdx.x*blockDim.x+threadIdx.x;
	const float x=scale(i,len);
	d_out[i]=distance(x,ref);
	printf("blockIdx = %2d, threadId = %2d, i = %2d: dist from %f to %f is %f.\n", 
		blockIdx.x, threadIdx.x, i, x,ref,d_out[i]);
}

int main()
{
  // Create an array of N floats (initialized to 0.0).
  // We will overwrite these values to store our results.

  // Choose a reference value from which distances are measured.
  const float ref = 0.5f;
	float *d_out=0;
	hipMalloc(&d_out, N*sizeof(float));

	distanceKernel<<<N/TPB, TPB>>>(d_out, ref,N);
	hipFree(d_out);
	

	
  return 0;
}
