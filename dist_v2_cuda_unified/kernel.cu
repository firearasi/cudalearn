
#include <hip/hip_runtime.h>
#include <iostream>
#define N 640000
#define TPB 32

__host__
float scale(int i, int n)
{
	return float(i) / ((float)(n-1));
}
__device__
float distance(float x1, float x2)
{
  return sqrt((x2 - x1)*(x2 - x1));
}


__global__ 
void distanceKernel(float *d_out, float *d_in, float ref)
{
    const int i = blockIdx.x*blockDim.x+threadIdx.x;
    float x=d_in[i];
    d_out[i]=distance(x,ref);
    //printf("blockIdx = %2d, threadId = %2d, i = %2d: dist from %f to %f is %f.\n",
    //    blockIdx.x, threadIdx.x, i, x,ref,d_out[i]);
}


int main()
{
	float *d_in = 0;
	float *d_out = 0;
	const float ref = 0.5;
	hipMallocManaged(&d_in, N*sizeof(float));
	hipMallocManaged(&d_out, N*sizeof(float));

	for(int i=0; i < N;i++)
		d_in[i]=scale(i,N);
		
	distanceKernel<<<N/TPB, TPB>>>(d_out, d_in, ref);
	hipDeviceSynchronize();
	hipFree(d_in);
	hipFree(d_out);
	return 0;
}



