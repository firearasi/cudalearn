
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 1000
#define TPB 32 // Threads per block



__global__ void  summationKernel(int *d_array, int n, int *d_res)
{
	const int idx=threadIdx.x+blockIdx.x*blockDim.x;
	const int s_idx=threadIdx.x;
	__shared__ int s_array[TPB];
	if(idx<n)
		s_array[s_idx]=d_array[idx];
	else
	{
		s_array[s_idx]=0;
		return;
	}
	__syncthreads();

	for(int s=blockDim.x/2;s>0;s>>=1)
	{
		if(s_idx<s)
		{
			s_array[s_idx]+=s_array[s_idx+s];
		}
	}
	__syncthreads();

	if(s_idx==0)
	{
		atomicAdd(d_res, s_array[0]);
	}
}

void summationLauncher(int *array, int n)
{
	int *d_array=0;
	hipMalloc(&d_array,n*sizeof(int));
	hipMemcpy(d_array,array,n*sizeof(int),hipMemcpyHostToDevice);
	int *d_res;
	hipMalloc(&d_res,sizeof(int));
	hipMemset(d_res,0,sizeof(int));
	int blocks = (N+TPB-1)/TPB;

	summationKernel<<<blocks,TPB>>>(d_array,n,d_res);
	int res;
	hipMemcpy(&res,d_res,sizeof(int), hipMemcpyDeviceToHost);
	printf("Sum is %d\n", res);
	hipFree(d_array);
	hipFree(d_res);
}




int main()
{
	int array[N];
	for(int i=0;i<N;i++)
	{
		array[i]=i;
	}
	summationLauncher(array,N);
	return 0;
}
